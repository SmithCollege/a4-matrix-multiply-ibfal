#include <iostream>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "hipblas.h"
 
#define TILE_WIDTH 5

double get_clock() {
  struct timeval tv; int ok;
  ok = gettimeofday(&tv, (void *) 0);
  if (ok<0) { printf("gettimeofday error"); }
  return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}
 
void gpu_blas( double* M,  double* N,  double* P, int width){
	const double a = 1;
	const double b = 0;
	const double *A = &a;
	const double *B = &b;

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,width,width, width, A,M, width,N,width, B, P, width);	

	hipblasDestroy(handle);
}
 
 int main(){
     int width = 100000;
     double *x,*y,*z, *dx, *dy, *dz;
 
     dx = (double *)malloc(sizeof(double) * width * width);
     dy = (double *)malloc(sizeof(double) * width * width);
     dz = (double *)malloc(sizeof(double) * width * width);
 
     hipMallocManaged(&x, sizeof(double) * width * width);
     hipMallocManaged(&y, sizeof(double) * width * width);
     hipMallocManaged(&z, sizeof(double) * width * width);
 
     for (int i = 0; i < width; i++) {
       for (int j = 0; j < width; j++) {
         dx[i * width + j] = 1.0; // x[i][j]
         dy[i * width + j] = 1.0;
       }
      }
 
    hipMemcpy(x, dx, sizeof(double) * width * width, hipMemcpyHostToDevice);
    hipMemcpy(y,dy, sizeof(double) * width * width, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

	double t0 = get_clock();
    gpu_blas(x, y, z, width);
	double t1 = get_clock();
	
    hipMemcpy(dz,z, sizeof(double) * width * width, hipMemcpyDeviceToHost);
          
	printf("\n");
	printf("Time: %f ns\n", (1000000000.0*(t1-t0)));
    printf("\n");
    
    for (int i = 0; i < width; i++) {       
    	for (int j = 0; j < width; j++) {
        	if (z[i * width + j] != width) {
            	 printf("Error at z[%d][%d]: %f\n", i, j, z[i * width + j]);
              }
            }
          }
    return 0;
}
